#include "hip/hip_runtime.h"
#include <optix_device.h>

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <raygBoundary.hpp>
#include <raygLaunchParams.hpp>
#include <raygPerRayData.hpp>
#include <raygRNG.hpp>
#include <raygReflection.hpp>
#include <raygSBTRecords.hpp>
#include <raygSource.hpp>

#include <models/psgPipelineParameters.hpp>

#include "CallableWrapper.cuh"

#include <vcContext.hpp>

using namespace viennaray::gpu;
using namespace viennacore;

extern "C" __constant__ viennaray::gpu::LaunchParams launchParams;

enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };

extern "C" __global__ void __intersection__() {
  const HitSBTDataDisk *sbtData =
      (const HitSBTDataDisk *)optixGetSbtDataPointer();
  PerRayData *prd = (PerRayData *)getPRD<PerRayData>();

  // Get the index of the AABB box that was hit
  const int primID = optixGetPrimitiveIndex();

  // Read geometric data from the primitive that is inside that AABB box
  const Vec3Df diskOrigin = sbtData->point[primID];
  const Vec3Df normal = sbtData->normal[primID];
  const float radius = sbtData->radius;

  bool valid = true;
  float prodOfDirections = DotProduct(normal, prd->dir);

  // Backface hits have to be reported so CH can let the ray through or kill the
  // ray if needed
  // valid &= DotProduct(prd->dir, normal) <= 0.0f;

  // Check if ray is not parallel to the plane
  valid &= fabsf(prodOfDirections) >= 1e-6f;

  float ddneg = DotProduct(diskOrigin, normal);
  float t = (ddneg - DotProduct(normal, prd->pos)) / prodOfDirections;
  // Avoid negative t or self intersections
  valid &= t > 1e-4f; // Maybe lower this further, but 1e-4f works for now

  const Vec3Df intersection = prd->pos + prd->dir * t;

  // Check if within disk radius
  const Vec3Df diff = intersection - diskOrigin;
  float distance = DotProduct(diff, diff);
  valid &= distance < radius * radius;

  if (valid) {
    // Collect all intersections and filter neighbors in CH shader
    if (!sbtData->base.isBoundary && prd->tempCount < MAX_NEIGHBORS) {
      prd->tValues[prd->tempCount] = t;
      prd->primIDs[prd->tempCount] = primID;
      prd->tempCount++;
    }

    // Has to pass a dummy t value so later intersections are not ignored
    optixReportIntersection(t + prd->tThreshold, 0);
  }
}

extern "C" __global__ void __closesthit__() {
  const HitSBTDataDisk *sbtData =
      (const HitSBTDataDisk *)optixGetSbtDataPointer();
  PerRayData *prd = (PerRayData *)getPRD<PerRayData>();

  const unsigned int primID = optixGetPrimitiveIndex();
  prd->tMin = optixGetRayTmax() - prd->tThreshold;
  prd->primID = primID;

  const Vec3Df normal = sbtData->normal[primID];

  // If closest hit was on backside, let it through once
  if (DotProduct(prd->dir, normal) > 0.0f) {
    // If back was hit a second time, kill the ray
    if (prd->hitFromBack) {
      prd->rayWeight = 0.f;
      return;
    }
    prd->hitFromBack = true;
    prd->pos = prd->pos + prd->tMin * prd->dir;
    return;
  }

  if (sbtData->base.isBoundary) {
    prd->numBoundaryHits++;
    // This is effectively the miss shader
    if (launchParams.D == 2 &&
        (primID == 2 || primID == 3)) { // bottom or top - ymin or ymax
      prd->rayWeight = 0.0f;
      return;
    }
    if (launchParams.D == 3 &&
        (primID == 4 || primID == 5)) { // bottom or top - zmin or zmax
      prd->rayWeight = 0.0f;
      return;
    }

    if (launchParams.periodicBoundary) {
      applyPeriodicBoundary(prd, sbtData, launchParams.D);
    } else {
      reflectFromBoundary(prd, sbtData, launchParams.D);
    }
  } else {
    // ------------- NEIGHBOR FILTERING --------------- //
    // Keep only hits close to tMin
    prd->ISCount = 0;
    for (int i = 0; i < prd->tempCount; ++i) {
      if (fabsf(prd->tValues[i] - prd->tMin) < prd->tThreshold &&
          prd->ISCount < MAX_NEIGHBORS) {
        prd->TIndex[prd->ISCount++] = prd->primIDs[i];
      }
    }

    // // CPU like neighbor detection
    // prd->ISCount = 0;
    // for (int i = 0; i < prd->tempCount; ++i) {
    //   float distance = viennacore::Distance(sbtData->point[primID],
    //                                         sbtData->point[prd->primIDs[i]]);
    //   if (distance < 2 * sbtData->radius && prd->ISCount < MAX_NEIGHBORS) {
    //     prd->TIndex[prd->ISCount++] = prd->primIDs[i];
    //   }
    // }

    // // Actual equivalent to CPU version
    // prd->TIndex[0] = primID;
    // prd->ISCount = 1;
    // for (int i = 0; i < launchParams.maxNeighbors; ++i) {
    //   int neighborIdx =
    //       launchParams.neighbors[primID * launchParams.maxNeighbors + i];
    //   if (neighborIdx == -1)
    //     continue;
    //   const Vec3Df diskOrigin = sbtData->point[neighborIdx];
    //   const Vec3Df normal = sbtData->normal[neighborIdx];
    //   const float radius = sbtData->radius;

    //   bool valid = true;
    //   float prodOfDirections = DotProduct(normal, prd->dir);
    //   // valid &= DotProduct(prd->dir, normal) <= 0.0f;
    //   valid &= fabsf(prodOfDirections) >= 1e-6f;

    //   float ddneg = DotProduct(diskOrigin, normal);
    //   float t = (ddneg - DotProduct(normal, prd->pos)) / prodOfDirections;
    //   valid &= t > 1e-4f;

    //   const Vec3Df intersection = prd->pos + prd->dir * t;
    //   const Vec3Df diff = intersection - diskOrigin;
    //   float distance = DotProduct(diff, diff);
    //   valid &= distance < radius * radius;
    //   if (valid)
    //     prd->TIndex[prd->ISCount++] = neighborIdx;
    // }

    // ------------- SURFACE COLLISION --------------- //
    unsigned callIdx;

    callIdx = callableIndex(launchParams.particleType, CallableSlot::COLLISION);
    optixDirectCall<void, const HitSBTDataDisk *, PerRayData *>(callIdx,
                                                                sbtData, prd);

    // ------------- REFLECTION --------------- //
    callIdx =
        callableIndex(launchParams.particleType, CallableSlot::REFLECTION);
    optixDirectCall<void, const HitSBTDataDisk *, PerRayData *>(callIdx,
                                                                sbtData, prd);
  }
}

extern "C" __global__ void __miss__() { getPRD<PerRayData>()->rayWeight = 0.f; }

extern "C" __global__ void __raygen__() {
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dims = optixGetLaunchDimensions();
  const int linearLaunchIndex =
      idx.x + idx.y * dims.x + idx.z * dims.x * dims.y;

  // per-ray data
  PerRayData prd;
  prd.tThreshold = 1.1f * launchParams.gridDelta;
  // prd.tThreshold = 0.f;
  // each ray has its own RNG state
  initializeRNGState(&prd, linearLaunchIndex, launchParams.seed);

  // initialize ray position and direction
  initializeRayPosition(&prd, &launchParams, launchParams.D);
  initializeRayDirection(&prd, launchParams.cosineExponent, launchParams.D);

  unsigned callIdx =
      callableIndex(launchParams.particleType, CallableSlot::INIT);
  optixDirectCall<void, const HitSBTDataDisk *, PerRayData *>(callIdx, nullptr,
                                                              &prd);

  // the values we store the PRD pointer in:
  uint32_t u0, u1;
  packPointer((void *)&prd, u0, u1);

  while (continueRay(launchParams, prd)) {
    optixTrace(launchParams.traversable, // traversable GAS
               make_float3(prd.pos[0], prd.pos[1], prd.pos[2]), // origin
               make_float3(prd.dir[0], prd.dir[1], prd.dir[2]), // direction
               1e-4f,                                           // tmin
               1e20f,                                           // tmax
               0.0f,                                            // rayTime
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,              // SBT offset
               RAY_TYPE_COUNT,                // SBT stride
               SURFACE_RAY_TYPE,              // missSBTIndex
               u0, u1);
    prd.tempCount = 0; // Reset PerRayData
  }
}
